#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include "GameOfLife.h"
#include "Methods.h"

void usage(){

   fprintf(stderr, "usage: ./life gameGridWidth gameGridHeight blockWidth blockHeight numOfPasses\n");
   fprintf(stderr, "usage: ./life\n");
}

int main (int argc, const char * argv[]) {
	// Create a grid (2d array of values) for Game of Life
	int gameGridHeight, gameGridWidth, blockHeight, blockWidth, numOfPasses;
	if (argc != 1 || argc != 6)
		usage();
	if (argc == 6){
		gameGridWidth = strToInt((char *)argv[1]);
		gameGridHeight = strToInt((char *)argv[2]);
		blockWidth = strToInt((char *)argv[3]);
		blockHeight = strToInt((char *)argv[4]);
		numOfPasses = strToInt((char *)argv[5]);
	}
	else{
		gameGridWidth = 32;
		gameGridHeight = 10;
		blockWidth = 32;
		blockHeight = 10;
		numOfPasses = 18;
	}
	int *gameGridIn = (int *) malloc(gameGridHeight * gameGridWidth * sizeof(int));
	bool pause = false;
	// Initialize the grid
	initializeArrays(gameGridIn, gameGridWidth, gameGridHeight);
	
	size_t pitch;

	// allocate device memory for data in
	int *d_gameGridIn;
	hipMallocPitch( (void**) &d_gameGridIn, &pitch, gameGridWidth * sizeof(int), gameGridHeight);
	
	// copy host memory to device memory for data in
	hipMemcpy2D( d_gameGridIn, pitch, gameGridIn, gameGridWidth * sizeof(int), gameGridWidth * sizeof(int), gameGridHeight, hipMemcpyHostToDevice);

	int gridWidth = (int) ceil( (gameGridWidth) / (float)blockWidth);
	int gridHeight = (int) ceil( (gameGridHeight) / (float)blockHeight);
	printf("block width: %d, block height: %d, grid width: %d, grid height: %d,\n\n", blockWidth, blockHeight, gridWidth, gridHeight);

	// Each block gets a shared memory region of this size.
	unsigned int shared_mem_size = ((blockWidth + 2) * (blockHeight+2)) * sizeof(int); 

	// Format the grid, which is a collection of blocks. 
   	dim3  grid( gridWidth, gridHeight, 1);
   
   	// Format the blocks. 
   	dim3  threads( blockWidth, blockHeight, 1);

	// When game is paused - allow the user to modify the grid values
	// When game is played - make the grid follow the rules
	printf("Starting grid:\n");	
	printArray(gameGridIn, gameGridHeight, gameGridWidth, 1);
	for (int i = 0; (i < numOfPasses) && (!pause); i++){
		//execute the kernel
		playGame<<< grid, threads, shared_mem_size >>>( d_gameGridIn, pitch/sizeof(int), gameGridWidth, gameGridHeight);
		//Print the array
		hipMemcpy2D( gameGridIn, gameGridWidth * sizeof(int), d_gameGridIn, pitch, gameGridWidth * sizeof(int), gameGridHeight, hipMemcpyDeviceToHost);
		printf("Grid Generation: %d\n", i+1);
		printArray(gameGridIn, gameGridHeight, gameGridWidth, 1);
	}

	hipDeviceSynchronize();
	
	hipMemcpy2D( gameGridIn, gameGridWidth * sizeof(int), d_gameGridIn, pitch, gameGridWidth * sizeof(int), gameGridHeight,hipMemcpyDeviceToHost);

}
